#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <histogram.h>
#include <cuda_gl_interop.h>

texture<uint8_t, 2, hipReadModeElementType> tex_y;
texture<uint8_t, 2, hipReadModeElementType> tex_u;
texture<uint8_t, 2, hipReadModeElementType> tex_v;

__device__ int dev_max[3];

__device__ inline uint8_t red(uint8_t y, uint8_t v)
{
    return (uint8_t) (y+1.5958*v);
}

__device__ inline uint8_t green(uint8_t y, uint8_t u, uint8_t v)
{
    return (uint8_t) (y-0.39173*u-0.81290*v);
}

__device__ inline uint8_t blue(uint8_t y, uint8_t u)
{
    return (uint8_t) (y+2.017*u);
}

__global__ void comp_histogram(GLint* hist_r, GLint* hist_g, GLint* hist_b, int w, int h)
{
    int x, y;
    uint8_t r_c, g_c, b_c;
    uint8_t y_c, u_c, v_c;

    __shared__ int temp_hist_r[256];
    __shared__ int temp_hist_g[256];
    __shared__ int temp_hist_b[256];
 
    if (blockIdx.x == 0)
    {
        hist_r[1+(2*threadIdx.x)] = 0;
        hist_g[1+(2*threadIdx.x)] = 0;
        hist_b[1+(2*threadIdx.x)] = 0;

        hist_r[(2*threadIdx.x)] = (threadIdx.x == 0)?0:threadIdx.x;
        hist_g[(2*threadIdx.x)] = (threadIdx.x == 0)?0:threadIdx.x;
        hist_b[(2*threadIdx.x)] = (threadIdx.x == 0)?0:threadIdx.x;
    }

    __syncthreads();

    temp_hist_r[threadIdx.x] = 0;
    temp_hist_g[threadIdx.x] = 0;
    temp_hist_b[threadIdx.x] = 0;

    __syncthreads();

    y = blockIdx.x;
    while (y < h)
    {
        x = threadIdx.x;
        while (x < w)
        {
            y_c = tex2D(tex_y, x, y);
            u_c = tex2D(tex_u, x/2, y/2);
            v_c = tex2D(tex_v, x/2, y/2);

            r_c = red(y_c, v_c);
            g_c = green(y_c, u_c, v_c);
            b_c = blue(y_c, u_c);

            atomicAdd(&temp_hist_r[r_c], 1);
            atomicAdd(&temp_hist_g[g_c], 1);
            atomicAdd(&temp_hist_b[b_c], 1);    

            x += blockDim.x;      
        }
        y += gridDim.x;
    }

    __syncthreads();

    atomicAdd(&hist_r[1+(2*threadIdx.x)], temp_hist_r[threadIdx.x]);
    atomicAdd(&hist_g[1+(2*threadIdx.x)], temp_hist_g[threadIdx.x]);
    atomicAdd(&hist_b[1+(2*threadIdx.x)], temp_hist_b[threadIdx.x]);
}

__global__ void get_max(GLint* hist_r, GLint* hist_g, GLint* hist_b/*, GLint* max*/)
{
    int i;
    GLint* hist;
    __shared__ GLint max_hist[3];

    hist = (threadIdx.x == 0)?hist_r:((threadIdx.x == 1)?hist_g:hist_b);
    max_hist[threadIdx.x] = 0;

    __syncthreads();
    
    for (i = 1; i < 256; i++)
    {
        if (hist[1+(2*i)] > max_hist[threadIdx.x])
        {
            max_hist[threadIdx.x] = hist[1+(2*i)];
        }
    }

    __syncthreads();

    dev_max[threadIdx.x] = max_hist[threadIdx.x];

    /*if (threadIdx.x == 0)
    {
        *max = 0;
        for (i = 0 ; i < 3; i++)
        {
            if (max_hist[i] > *max)
            {
                *max = max_hist[i];
            }
        }
    }*/
}

void print_cuda_device_info();

void compute_histogram(unsigned int* texture, unsigned int* hist_obj, int* hist_max, int width, int height)
{
    GLint* dev_hist_r = 0;
    GLint* dev_hist_g = 0;
    GLint* dev_hist_b = 0;

    cudaGLRegisterBufferObject(hist_obj[0]);
    cudaGLMapBufferObject((void **)&dev_hist_r, hist_obj[0]);
    cudaGLRegisterBufferObject(hist_obj[1]);
    cudaGLMapBufferObject((void **)&dev_hist_g, hist_obj[1]);
    cudaGLRegisterBufferObject(hist_obj[2]);
    cudaGLMapBufferObject((void **)&dev_hist_b, hist_obj[2]);

    hipError_t err;
    hipGraphicsResource* res[3] = {0, 0, 0};
    err = hipGraphicsGLRegisterImage(&res[0], texture[0], GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly);
    if (err != hipSuccess)
    {
        printf("hipGraphicsGLRegisterImage Failed: %s", hipGetErrorString(hipGetLastError()));
        exit(0);
    }
    err = hipGraphicsGLRegisterImage(&res[1], texture[1], GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly);
    if (err != hipSuccess)
    {
        printf("hipGraphicsGLRegisterImage Failed: %s", hipGetErrorString(hipGetLastError()));
        exit(0);
    }
    err = hipGraphicsGLRegisterImage(&res[2], texture[2], GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly);
    if (err != hipSuccess)
    {
        printf("hipGraphicsGLRegisterImage Failed: %s", hipGetErrorString(hipGetLastError()));
        exit(0);
    }

    hipGraphicsMapResources(3, res);

    hipArray* y_array = 0;
    hipArray* u_array = 0;
    hipArray* v_array = 0;
    err = hipGraphicsSubResourceGetMappedArray(&y_array, res[0], 0, 0);
    if (err != hipSuccess)
    {
        printf("hipGraphicsSubResourceGetMappedArray Failed: %s", hipGetErrorString(hipGetLastError()));
        exit(0);
    }
    err = hipGraphicsSubResourceGetMappedArray(&u_array, res[1], 0, 0);
    if (err != hipSuccess)
    {
        printf("hipGraphicsSubResourceGetMappedArray Failed: %s", hipGetErrorString(hipGetLastError()));
        exit(0);
    }
    err = hipGraphicsSubResourceGetMappedArray(&v_array, res[2], 0, 0);
    if (err != hipSuccess)
    {
        printf("hipGraphicsSubResourceGetMappedArray Failed: %s", hipGetErrorString(hipGetLastError()));
        exit(0);
    }

    hipChannelFormatDesc y_chan_desc = hipCreateChannelDesc<uint8_t>();
    hipChannelFormatDesc u_chan_desc = hipCreateChannelDesc<uint8_t>();
    hipChannelFormatDesc v_chan_desc = hipCreateChannelDesc<uint8_t>();
	if (hipBindTextureToArray(&tex_y, y_array, &y_chan_desc) != hipSuccess) {
		printf("Failed to bind y texture: %s\n", hipGetErrorString(hipGetLastError()));
		exit(0);
	}
	if (hipBindTextureToArray(&tex_u, u_array, &u_chan_desc) != hipSuccess) {
		printf("Failed to bind y texture: %s\n", hipGetErrorString(hipGetLastError()));
		exit(0);
	}
	if (hipBindTextureToArray(&tex_v, v_array, &v_chan_desc) != hipSuccess) {
		printf("Failed to bind y texture: %s\n", hipGetErrorString(hipGetLastError()));
		exit(0);
	}

    comp_histogram<<<64, 256>>>(dev_hist_r, dev_hist_g, dev_hist_b, width, height);
    hipDeviceSynchronize();
    get_max<<<1, 3>>>(dev_hist_r, dev_hist_g, dev_hist_b);

    hipMemcpyFromSymbol(hist_max, HIP_SYMBOL(dev_max), 3*sizeof(int));

    hipUnbindTexture(&tex_y);
    hipUnbindTexture(&tex_u);
    hipUnbindTexture(&tex_v);

    cudaGLUnmapBufferObject(hist_obj[0]);
    cudaGLUnmapBufferObject(hist_obj[1]);
    cudaGLUnmapBufferObject(hist_obj[2]);

    cudaGLUnregisterBufferObject(hist_obj[0]);
    cudaGLUnregisterBufferObject(hist_obj[1]);
    cudaGLUnregisterBufferObject(hist_obj[2]);

    hipGraphicsUnmapResources(3, res);
    hipGraphicsUnregisterResource(res[0]);
    hipGraphicsUnregisterResource(res[1]);
    hipGraphicsUnregisterResource(res[2]);
}

void print_cuda_device_info()
{
    int count = 0;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count);

    printf("\nCUDA Device Count: %d", count); 
    for (int i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        printf("\nDevice: %d", i);
        printf("\nName: %s", prop.name);
        printf("\nRevision: Major: %d, Minor: %d", prop.major, prop.minor);
        printf("\nWarp Size: %d", prop.warpSize);
        printf("\nMemory Bus width: %d", prop.memoryBusWidth);
        printf("\nMemory Clock Rate: %d", prop.memoryClockRate);
        printf("\nConcurrent Kernels: %d", prop.concurrentKernels);
        printf("\nMultiprocessor count: %d", prop.multiProcessorCount);
        printf("\nTotal Global Memory: %d", (int)prop.totalGlobalMem);
        printf("\nTotal Constant Memory: %d", (int)prop.totalConstMem);
        printf("\nShared Memory per Block: %d", (int)prop.sharedMemPerBlock);
        printf("\nMax grid dimensions: (%d, %d, %d)", prop.maxGridSize[0], 
                                                       prop.maxGridSize[1], 
                                                       prop.maxGridSize[2]);
        printf("\nMax threads per block: %d", prop.maxThreadsPerBlock);
        printf("\nMax threads dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], 
                                                       prop.maxThreadsDim[1], 
                                                       prop.maxThreadsDim[2]);
    }
}

