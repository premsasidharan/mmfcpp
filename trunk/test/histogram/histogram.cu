#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/*
red = y+1.5958*v;
green = y-0.39173*u-0.81290*v;
blue = y+2.017*u;*/

texture<uint8_t, 2, hipReadModeElementType> tex_y;
texture<uint8_t, 2, hipReadModeElementType> tex_u;
texture<uint8_t, 2, hipReadModeElementType> tex_v;

__device__ inline uint8_t red(uint8_t y, uint8_t v)
{
    return (uint8_t) (y+1.5958*v);
}

__device__ inline uint8_t green(uint8_t y, uint8_t u, uint8_t v)
{
    return (uint8_t) (y-0.39173*u-0.81290*v);
}

__device__ inline uint8_t blue(uint8_t y, uint8_t u)
{
    return (uint8_t) (y+2.017*u);
}

__global__ void comp_histogram(uint32_t* hist_r, uint32_t* hist_g, uint32_t* hist_b, int w, int h)
{
    int x, y;
    uint8_t r_c, g_c, b_c;
    uint8_t y_c, u_c, v_c;

    __shared__ uint32_t temp_hist_r[256];
    __shared__ uint32_t temp_hist_g[256];
    __shared__ uint32_t temp_hist_b[256];


    temp_hist_r[threadIdx.x] = 0;
    temp_hist_g[threadIdx.x] = 0;
    temp_hist_b[threadIdx.x] = 0;

    __syncthreads();

    y = blockIdx.x;
    while (y < h)
    {
        x = threadIdx.x;
        while (x < w)
        {
            y_c = tex2D(tex_y, x, y);
            u_c = tex2D(tex_u, x/2, y/2);
            v_c = tex2D(tex_v, x/2, y/2);

            r_c = red(y_c, v_c);
            g_c = green(y_c, u_c, v_c);
            b_c = blue(y_c, u_c);

            atomicAdd(&temp_hist_r[r_c], 1);
            atomicAdd(&temp_hist_g[g_c], 1);
            atomicAdd(&temp_hist_b[b_c], 1);    

            x += blockDim.x;      
        }
        y += gridDim.x;
    }

    __syncthreads();

    atomicAdd(&hist_r[threadIdx.x], temp_hist_r[threadIdx.x]);
    atomicAdd(&hist_g[threadIdx.x], temp_hist_g[threadIdx.x]);
    atomicAdd(&hist_b[threadIdx.x], temp_hist_b[threadIdx.x]);
}

void print_cuda_device_info();

int main(int argc, char** argv)
{
    uint32_t host_hist_r[256];
    uint32_t host_hist_g[256];
    uint32_t host_hist_b[256];

    if (argc != 4)
    {
        printf("\nFailed !!!\n");
        return 0;
    }

	int width = atoi(argv[2]);
	int height = atoi(argv[3]);
	const char* file_path = argv[1];
	
    hipArray* dev_src = 0;
    hipArray* dev_src_u = 0;
    hipArray* dev_src_v = 0;

    uint32_t* dev_hist_r = 0;
    uint32_t* dev_hist_g = 0;
    uint32_t* dev_hist_b = 0;
 
    int y_size = width*height;
    int uv_size = (y_size>>2);
	int yuv_size = y_size+(uv_size<<1);
    uint8_t* host_src_mem = new uint8_t[yuv_size];

    hipChannelFormatDesc y_chan_desc = hipCreateChannelDesc<uint8_t>();	
    hipChannelFormatDesc u_chan_desc = hipCreateChannelDesc<uint8_t>();	
    hipChannelFormatDesc v_chan_desc = hipCreateChannelDesc<uint8_t>();

    hipMallocArray(&dev_src, &y_chan_desc, width, height);
    hipMallocArray(&dev_src_u, &u_chan_desc, width>>1, height>>1);
    hipMallocArray(&dev_src_v, &v_chan_desc, width>>1, height>>1);

    hipMalloc((void **)&dev_hist_r, 256*sizeof(uint32_t));
    hipMalloc((void **)&dev_hist_g, 256*sizeof(uint32_t));
    hipMalloc((void **)&dev_hist_b, 256*sizeof(uint32_t));

    tex_y.addressMode[0] = hipAddressModeWrap;
    tex_y.addressMode[1] = hipAddressModeWrap;
    tex_y.filterMode = hipFilterModePoint;
    tex_y.normalized = false;

    if (hipBindTextureToArray(&tex_y, dev_src, &y_chan_desc) != hipSuccess) {
	    printf("Failed to bind y texture: %s\n", hipGetErrorString(hipGetLastError()));
	    return 0;
    }

    tex_u.addressMode[0] = hipAddressModeWrap;
    tex_u.addressMode[1] = hipAddressModeWrap;
    tex_u.filterMode = hipFilterModePoint;
    tex_u.normalized = false;

    if (hipBindTextureToArray(&tex_u, dev_src_u, &u_chan_desc) != hipSuccess) {
	    printf("Failed to bind u texture: %s\n", hipGetErrorString(hipGetLastError()));
	    return 0;
    }

    tex_v.addressMode[0] = hipAddressModeWrap;
    tex_v.addressMode[1] = hipAddressModeWrap;
    tex_v.filterMode = hipFilterModePoint;
    tex_v.normalized = false;

    if (hipBindTextureToArray(&tex_v, dev_src_v, &v_chan_desc) != hipSuccess) {
	    printf("Failed to bind v texture: %s\n", hipGetErrorString(hipGetLastError()));
	    return 0;
    }

    int frame = 0;
	FILE* file = fopen(file_path, "r");
	
    while (1)
    {
	    if (0 == fread(host_src_mem, 1, yuv_size, file))
        {
            break;
        }

        if (hipMemcpyToArray(dev_src, 0, 0, host_src_mem, y_size, hipMemcpyHostToDevice) != hipSuccess)
        {
		    printf("1 Failed hipMemcpyToArray: %s\n", hipGetErrorString(hipGetLastError()));
		    return 0;
        }
        if (hipMemcpyToArray(dev_src_u, 0, 0, &host_src_mem[y_size], uv_size, hipMemcpyHostToDevice) != hipSuccess)
        {
		    printf("2 Failed hipMemcpyToArray: %s\n", hipGetErrorString(hipGetLastError()));
		    return 0;
        }
        if (hipMemcpyToArray(dev_src_v, 0, 0, &host_src_mem[y_size+uv_size], uv_size, hipMemcpyHostToDevice) != hipSuccess)
        {
		    printf("3 Failed hipMemcpyToArray: %s\n", hipGetErrorString(hipGetLastError()));
		    return 0;
        }

        hipMemset(dev_hist_r, 0, 256*sizeof(uint32_t));
        hipMemset(dev_hist_g, 0, 256*sizeof(uint32_t));
        hipMemset(dev_hist_b, 0, 256*sizeof(uint32_t));

        comp_histogram<<<64, 256>>>(dev_hist_r, dev_hist_g, dev_hist_b, width, height);

        hipDeviceSynchronize();

        hipMemcpy(host_hist_r, dev_hist_r, 256*sizeof(uint32_t), hipMemcpyDeviceToHost);
        hipMemcpy(host_hist_g, dev_hist_g, 256*sizeof(uint32_t), hipMemcpyDeviceToHost);
        hipMemcpy(host_hist_b, dev_hist_b, 256*sizeof(uint32_t), hipMemcpyDeviceToHost);

        printf("\nFrame: %d ", frame++);
        for (int i = 0; i < 256; i++)
        {
            printf("%d:(%u %u %u), ", i, host_hist_r[i], host_hist_g[i], host_hist_b[i]);
        }
    }
    fclose(file);
        
    hipUnbindTexture(&tex_v);
    hipUnbindTexture(&tex_u);
    hipUnbindTexture(&tex_y);

    hipFree(dev_hist_r); dev_hist_r = 0; 
    hipFree(dev_hist_g); dev_hist_g = 0; 
    hipFree(dev_hist_b); dev_hist_b = 0;  
 
    hipFree(dev_src_v); dev_src_v = 0;  
    hipFree(dev_src_u); dev_src_u = 0;   
    hipFree(dev_src); dev_src = 0;
    
    delete [] host_src_mem;
    host_src_mem = 0;

    return 0;
}

void print_cuda_device_info()
{
    int count = 0;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count);

    printf("\nCUDA Device Count: %d", count); 
    for (int i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        printf("\nDevice: %d", i);
        printf("\nName: %s", prop.name);
        printf("\nRevision: Major: %d, Minor: %d", prop.major, prop.minor);
        printf("\nWarp Size: %d", prop.warpSize);
        printf("\nMemory Bus width: %d", prop.memoryBusWidth);
        printf("\nMemory Clock Rate: %d", prop.memoryClockRate);
        printf("\nConcurrent Kernels: %d", prop.concurrentKernels);
        printf("\nMultiprocessor count: %d", prop.multiProcessorCount);
        printf("\nTotal Global Memory: %d", (int)prop.totalGlobalMem);
        printf("\nTotal Constant Memory: %d", (int)prop.totalConstMem);
        printf("\nShared Memory per Block: %d", (int)prop.sharedMemPerBlock);
        printf("\nMax grid dimensions: (%d, %d, %d)", prop.maxGridSize[0], 
                                                       prop.maxGridSize[1], 
                                                       prop.maxGridSize[2]);
        printf("\nMax threads per block: %d", prop.maxThreadsPerBlock);
        printf("\nMax threads dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], 
                                                       prop.maxThreadsDim[1], 
                                                       prop.maxThreadsDim[2]);
    }
}

