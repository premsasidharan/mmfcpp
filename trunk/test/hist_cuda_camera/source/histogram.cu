#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <histogram.h>
#include <cuda_gl_interop.h>

texture<uchar2, 2, hipReadModeElementType> tex_y;
texture<uchar4, 2, hipReadModeElementType> tex_uv;

__device__ int dev_max[3];

__device__ inline uint8_t red(uint8_t y, uint8_t v)
{
    return (uint8_t) (y+1.5958*v);
}

__device__ inline uint8_t green(uint8_t y, uint8_t u, uint8_t v)
{
    return (uint8_t) (y-0.39173*u-0.81290*v);
}

__device__ inline uint8_t blue(uint8_t y, uint8_t u)
{
    return (uint8_t) (y+2.017*u);
}

__global__ void comp_histogram(GLint* hist_r, GLint* hist_g, GLint* hist_b, int w, int h)
{
    int x, y;
    uint8_t r_c, g_c, b_c;
    uint8_t y_c, u_c, v_c;

    __shared__ int temp_hist_r[256];
    __shared__ int temp_hist_g[256];
    __shared__ int temp_hist_b[256];
 
    if (blockIdx.x == 0)
    {
        hist_r[1+(2*threadIdx.x)] = 0;
        hist_g[1+(2*threadIdx.x)] = 0;
        hist_b[1+(2*threadIdx.x)] = 0;

        hist_r[(2*threadIdx.x)] = (threadIdx.x == 0)?0:threadIdx.x;
        hist_g[(2*threadIdx.x)] = (threadIdx.x == 0)?0:threadIdx.x;
        hist_b[(2*threadIdx.x)] = (threadIdx.x == 0)?0:threadIdx.x;
    }

    __syncthreads();

    temp_hist_r[threadIdx.x] = 0;
    temp_hist_g[threadIdx.x] = 0;
    temp_hist_b[threadIdx.x] = 0;

    __syncthreads();

    y = blockIdx.x;
    while (y < h)
    {
        x = threadIdx.x;
        while (x < w)
        {
            y_c = tex2D(tex_y, x, y).x;
            u_c = tex2D(tex_uv, x/2, y).y;
            v_c = tex2D(tex_uv, x/2, y).w;

            r_c = red(y_c, v_c);
            g_c = green(y_c, u_c, v_c);
            b_c = blue(y_c, u_c);

            atomicAdd(&temp_hist_r[r_c], 1);
            atomicAdd(&temp_hist_g[g_c], 1);
            atomicAdd(&temp_hist_b[b_c], 1);    

            x += blockDim.x;      
        }
        y += gridDim.x;
    }

    __syncthreads();

    atomicAdd(&hist_r[1+(2*threadIdx.x)], temp_hist_r[threadIdx.x]);
    atomicAdd(&hist_g[1+(2*threadIdx.x)], temp_hist_g[threadIdx.x]);
    atomicAdd(&hist_b[1+(2*threadIdx.x)], temp_hist_b[threadIdx.x]);
}

__global__ void get_max(GLint* hist_r, GLint* hist_g, GLint* hist_b)
{
    int i;
    GLint* hist;
    __shared__ GLint max_hist[3];

    hist = (threadIdx.x == 0)?hist_r:((threadIdx.x == 1)?hist_g:hist_b);
    max_hist[threadIdx.x] = 0;

    __syncthreads();
    
    for (i = 1; i < 256; i++)
    {
        if (hist[1+(2*i)] > max_hist[threadIdx.x])
        {
            max_hist[threadIdx.x] = hist[1+(2*i)];
        }
    }

    __syncthreads();

    dev_max[threadIdx.x] = max_hist[threadIdx.x];
}

void print_cuda_device_info();

void compute_histogram(unsigned int* texture, unsigned int* hist_obj, int* hist_max, int width, int height)
{
    hipError_t err;
    GLint* dev_hist[3] = {0, 0, 0};
    hipArray* array[3] = {0, 0, 0};
    hipGraphicsResource* res[3] = {0, 0, 0};

    for (int i = 0; i < 3; i++)
    {
        cudaGLRegisterBufferObject(hist_obj[i]);
        cudaGLMapBufferObject((void **)&dev_hist[i], hist_obj[i]);
    }

    err = hipGraphicsGLRegisterImage(&res[0], texture[0], GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly);
    if (err != hipSuccess)
    {
        printf("hipGraphicsGLRegisterImage Failed: %s", hipGetErrorString(hipGetLastError()));
        exit(0);
    }

    err = hipGraphicsGLRegisterImage(&res[1], texture[1], GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly);
    if (err != hipSuccess)
    {
        printf("hipGraphicsGLRegisterImage Failed: %s", hipGetErrorString(hipGetLastError()));
        exit(0);
    }

    hipGraphicsMapResources(2, res);
    for (int i = 0; i < 2; i++)
    {
        err = hipGraphicsSubResourceGetMappedArray(&array[i], res[i], 0, 0);
        if (err != hipSuccess)
        {
            printf("hipGraphicsSubResourceGetMappedArray Failed: %s", hipGetErrorString(hipGetLastError()));
            exit(0);
        }
    }

    hipChannelFormatDesc y_chan_desc = hipCreateChannelDesc<uchar2>();
    if (hipBindTextureToArray(&tex_y, array[0], &y_chan_desc) != hipSuccess) {
	    printf("Failed to bind y texture: %s\n", hipGetErrorString(hipGetLastError()));
	    exit(0);
    }
    hipChannelFormatDesc uv_chan_desc = hipCreateChannelDesc<uchar4>();
    if (hipBindTextureToArray(&tex_uv, array[1], &uv_chan_desc) != hipSuccess) {
	    printf("Failed to bind uv texture: %s\n", hipGetErrorString(hipGetLastError()));
	    exit(0);
    }

    comp_histogram<<<64, 256>>>(dev_hist[0], dev_hist[1], dev_hist[2], width, height);
    hipDeviceSynchronize();
    get_max<<<1, 3>>>(dev_hist[0], dev_hist[1], dev_hist[2]);

    hipMemcpyFromSymbol(hist_max, HIP_SYMBOL(dev_max), 3*sizeof(int));

    hipUnbindTexture(&tex_y);
    hipUnbindTexture(&tex_uv);

    hipGraphicsUnmapResources(2, res);

    hipGraphicsUnregisterResource(res[0]);
    hipGraphicsUnregisterResource(res[1]);
    
    for (int i = 0; i < 3; i++)
    {
        cudaGLUnmapBufferObject(hist_obj[i]);
        cudaGLUnregisterBufferObject(hist_obj[i]);
    }
}

void print_cuda_device_info()
{
    int count = 0;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count);

    printf("\nCUDA Device Count: %d", count); 
    for (int i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        printf("\nDevice: %d", i);
        printf("\nName: %s", prop.name);
        printf("\nRevision: Major: %d, Minor: %d", prop.major, prop.minor);
        printf("\nWarp Size: %d", prop.warpSize);
        printf("\nMemory Bus width: %d", prop.memoryBusWidth);
        printf("\nMemory Clock Rate: %d", prop.memoryClockRate);
        printf("\nConcurrent Kernels: %d", prop.concurrentKernels);
        printf("\nMultiprocessor count: %d", prop.multiProcessorCount);
        printf("\nTotal Global Memory: %d", (int)prop.totalGlobalMem);
        printf("\nTotal Constant Memory: %d", (int)prop.totalConstMem);
        printf("\nShared Memory per Block: %d", (int)prop.sharedMemPerBlock);
        printf("\nMax grid dimensions: (%d, %d, %d)", prop.maxGridSize[0], 
                                                       prop.maxGridSize[1], 
                                                       prop.maxGridSize[2]);
        printf("\nMax threads per block: %d", prop.maxThreadsPerBlock);
        printf("\nMax threads dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], 
                                                       prop.maxThreadsDim[1], 
                                                       prop.maxThreadsDim[2]);
    }
}

